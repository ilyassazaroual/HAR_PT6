#include "hip/hip_runtime.h"
#include "imageProcessing.h"
#include "globalVars.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"

__global__
void myKernel(unsigned char *out,unsigned char *im, long int taille){
 
    int idx =blockIdx.x * blockDim.x +threadIdx.x;
        
	int reduceVal=50;
	
	if (idx <taille) {
		if(im[idx]>reduceVal){
			out[idx]=im[idx]/2;
		}else{
      			out[idx] =0;
		}
    } 	
}
/*
void filterImage_Cuda(unsigned char *out, unsigned char *im, int im_step, int im_cols, int im_rows){
double tstart, tend;
//cudaPrintfInit();
dim3 threads(1,2,3);
dim3 blocks(1,1,4);

unsigned char *cout, *cim;

hipSetDevice(0);
hipMalloc((void **)&cout, sizeof(unsigned char)*(im_cols*im_rows));
hipMalloc((void **)&cim, sizeof(unsigned char)*(im_cols*im_rows));

hipMemcpy(cout, out, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyHostToDevice);
hipMemcpy(cim, im, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyHostToDevice);

tstart = wallclock();
printf("We going to the kernel \n" );
printf(" %d \n and %d \n",im_cols,im_rows);
kernel<<< blocks, threads >>>(cout,cim,(im_cols*im_rows));
tend = wallclock();
printf("Time for kernel call is :%f milliseconds \n" , (tend-tstart)*1000.0 );

hipMemcpy(out, cout, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyDeviceToHost);
hipMemcpy(im, cim, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyDeviceToHost);

hipFree(cout);
hipFree(cim);

}
*/

/* 
 * Filter before matching 
 
void filterImage_Cuda(unsigned char *out, unsigned char *im, int im_step, int im_cols, int im_rows){
  if (!im || !out) return;
  ///// TO BE FILLED

#pragma acc kernels pcopyin(im[0:im_cols*im_rows]) pcopyout(out[0:im_cols*im_rows]) 
  {
#pragma acc loop independent
    for (int idx = 0; idx < im_cols*im_rows; idx++){
      int reduceVal = 50;
      if (im[idx] > reduceVal) out[idx] = im[idx] - reduceVal; 
      else out[idx] = 0;
    }
  }
}

*/
void filterImage(unsigned char *out, unsigned char *im, int im_step, int im_cols, int im_rows){
  if (!im || !out) return;

  for(int i = 3;i < im_rows-3;i++){
    for(int j = 3;j < im_cols-3;j++){
      double v1 = (2047.0 *(im[INDXs(im_step,i,j+1)] - im[INDXs(im_step,i,j-1)])
		   +913.0 *(im[INDXs(im_step,i,j+2)] - im[INDXs(im_step,i,j-2)])
		   +112.0 *(im[INDXs(im_step,i,j+3)] - im[INDXs(im_step,i,j-3)]))/8418.0;
      //v1 is not in the range NEED FIXING
      out[INDXs(im_step,i,j)] = v1;
    }
  }
}

void filterImage(float *out, float *im, int im_step, int im_cols, int im_rows){
  if (!im || !out) return;

  for(int i = 3;i < im_rows-3;i++){
    for(int j = 3;j < im_cols-3;j++){
      double v1 = (2047.0 *(im[INDXs(im_step,i,j+1)] - im[INDXs(im_step,i,j-1)])
                   +913.0 *(im[INDXs(im_step,i,j+2)] - im[INDXs(im_step,i,j-2)])
                   +112.0 *(im[INDXs(im_step,i,j+3)] - im[INDXs(im_step,i,j-3)]))/8418.0;
      //v1 is not in the range NEED FIXING                                                                                        
      out[INDXs(im_step,i,j)] = v1;
    }
  }
}
 
cv::Mat *filterImage(cv::Mat *image){

  cv::Mat *filtered = NULL;
  if (!image) return filtered;
  //Deep copy of the original                                                                                                                                                                                                         
  filtered = new cv::Mat(image->clone());
  unsigned char *fil = (unsigned char*)(filtered->data);

  unsigned char *im = (unsigned char*)(image->data);
  int im_step = image->step;
  int im_cols = image->cols;
  int im_rows = image->rows;
	 double tstart, tend;
	//cudaPrintfInit();
	dim3 threads(im_cols);
 	dim3 blocks(im_rows);

	unsigned char *cout,*cim;

	hipSetDevice(0);
	hipMalloc((void **)&cout, sizeof(unsigned char)*(im_cols*im_rows));
	hipMalloc((void **)&cim, sizeof(unsigned char)*(im_cols*im_rows));
	
	hipMemcpy(cim,im, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyHostToDevice);
	hipMemcpy(cout,fil, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyHostToDevice);

	printf("We going to the kernel \n" );
	printf(" %d \n and %d \n",im_cols,im_rows);
	
	tstart = wallclock();
	myKernel<<< blocks, threads >>>(cout,cim,(im_cols*im_rows));
	tend = wallclock();

	printf("Time for kernel call is :%f milliseconds \n" , (tend-tstart)*1000.0 );

	hipMemcpy(fil, cout, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyDeviceToHost);
	//hipMemcpy(cim, im, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyDeviceToHost);
	hipFree(cout);
	hipFree(cim);
  
	filtered->data=fil;
	//filterImage_Cuda(fil,im,im_step,im_cols,im_rows);
        return filtered;
}


void  filterImages(){
  for (int i=0; i< nbImages; i++){
    image_gray[i] = *filterImage(&image_gray[i]);
  }
}

void filterSamples(){
  for (int i=0; i< nbSamples; i++){
    sample_gray[i] = *filterImage(&sample_gray[i]);
  }
}
